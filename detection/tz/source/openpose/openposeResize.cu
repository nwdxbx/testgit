#include "hip/hip_runtime.h"
#include "openposeResize.hpp"

const auto THREADS_PER_BLOCK_1D = 16u;
const auto CUDA_NUM_THREADS = 512u;

template<typename T>
inline __device__ T fastMax(const T a, const T b)
{
    return (a > b ? a : b);
}

template<typename T>
inline __device__ T fastMin(const T a, const T b)
{
    return (a < b ? a : b);
}

template<class T>
inline __device__ T fastTruncate(T value, T min = 0, T max = 1)
{
    return fastMin(max, fastMax(min, value));
}


inline unsigned int getNumberCudaBlocks(const unsigned int totalRequired, const unsigned int numberCudaThreads = CUDA_NUM_THREADS)
{
    return (totalRequired + numberCudaThreads - 1) / numberCudaThreads;
}


template <typename T>
inline __device__ void cubicSequentialData(int* xIntArray, int* yIntArray, T& dx, T& dy, const T xSource, const T ySource, const int width, const int height)
{
    xIntArray[1] = fastTruncate(int(xSource + 1e-5), 0, width - 1);
    xIntArray[0] = fastMax(0, xIntArray[1] - 1);
    xIntArray[2] = fastMin(width - 1, xIntArray[1] + 1);
    xIntArray[3] = fastMin(width - 1, xIntArray[2] + 1);
    dx = xSource - xIntArray[1];

    yIntArray[1] = fastTruncate(int(ySource + 1e-5), 0, height - 1);
    yIntArray[0] = fastMax(0, yIntArray[1] - 1);
    yIntArray[2] = fastMin(height - 1, yIntArray[1] + 1);
    yIntArray[3] = fastMin(height - 1, yIntArray[2] + 1);
    dy = ySource - yIntArray[1];
}

template <typename T>
inline __device__ T cubicInterpolation(const T v0, const T v1, const T v2, const T v3, const T dx)
{
    // http://www.paulinternet.nl/?page=bicubic
    // const auto a = (-0.5f * v0 + 1.5f * v1 - 1.5f * v2 + 0.5f * v3);
    // const auto b = (v0 - 2.5f * v1 + 2.0 * v2 - 0.5 * v3);
    // const auto c = (-0.5f * v0 + 0.5f * v2);
    // out = ((a * dx + b) * dx + c) * dx + v1;
    return (-0.5f * v0 + 1.5f * v1 - 1.5f * v2 + 0.5f * v3) * dx * dx * dx
            + (v0 - 2.5f * v1 + 2.f * v2 - 0.5f * v3) * dx * dx
            - 0.5f * (v0 - v2) * dx // + (-0.5f * v0 + 0.5f * v2) * dx
            + v1;
}

template <typename T>
inline __device__ T cubicResize(const T* const sourcePtr, const T xSource, const T ySource, const int widthSource, const int heightSource, const int widthSourcePtr)
{
    int xIntArray[4];
    int yIntArray[4];
    T dx;
    T dy;
    cubicSequentialData(xIntArray, yIntArray, dx, dy, xSource, ySource, widthSource, heightSource);

    T temp[4];
    for (unsigned char i = 0; i < 4; i++)
    {
        const int offset = yIntArray[i]*widthSourcePtr;
        temp[i] = cubicInterpolation(sourcePtr[offset + xIntArray[0]], sourcePtr[offset + xIntArray[1]], sourcePtr[offset + xIntArray[2]], sourcePtr[offset + xIntArray[3]], dx);
    }
    return cubicInterpolation(temp[0], temp[1], temp[2], temp[3], dy);
}

template <typename T>
__global__ void resizeKernel(T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth, const int targetHeight)
{
    const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < targetWidth && y < targetHeight)
    {
        const auto scaleWidth = targetWidth / T(sourceWidth);
        const auto scaleHeight = targetHeight / T(sourceHeight);
        const T xSource = (x + 0.5f) / scaleWidth - 0.5f;
        const T ySource = (y + 0.5f) / scaleHeight - 0.5f;

        targetPtr[y*targetWidth+x] = cubicResize(sourcePtr, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
    }
}

template <typename T>
void resizeAndMergeGpu(T* targetPtr, const T* const sourcePtr, const std::array<int, 4>& targetSize, const std::array<int, 4>& sourceSize)
{
    const auto num = sourceSize[0];
    const auto channels = sourceSize[1];
    const auto sourceHeight = sourceSize[2];
    const auto sourceWidth = sourceSize[3];
    const auto targetHeight = targetSize[2];
    const auto targetWidth = targetSize[3];

    const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
    const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x), getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
    const auto sourceChannelOffset = sourceHeight * sourceWidth;
    const auto targetChannelOffset = targetWidth * targetHeight;

    for (auto n = 0; n < num; n++)
        for (auto c = 0; c < channels; c++)
            resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + (n*channels + c) * targetChannelOffset, sourcePtr + (n*channels + c) * sourceChannelOffset,
                    sourceWidth, sourceHeight, targetWidth, targetHeight);
}

template void resizeAndMergeGpu(float* targetPtr, const float* const sourcePtr, const std::array<int, 4>& targetSize, const std::array<int, 4>& sourceSize);
template void resizeAndMergeGpu(double* targetPtr, const double* const sourcePtr, const std::array<int, 4>& targetSize, const std::array<int, 4>& sourceSize);
